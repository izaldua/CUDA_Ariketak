
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 16 //Hau temporala da eta aldatu edo kendu egin behar da, tam aldagaiak aldatzen ditu erroreak ez agertzeko kodetzeko orduan


__global__ void GPU_eragiketak(float *A, float *B, float *C, float *D, int tam1,  int tam2, int tam3){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < tam1*tam3){
        float tmp_sum = 0.0f;
        int x = i / tam3;
        int y = i % tam3;
        for (int k = 0; k < tam2; k++){
            tmp_sum += A[x*tam2+k] * B[k*tam3+y];
            //D[x*tam3+y] = A[x*tam2+k] * B[k*tam3+y];
        }
        D[x*tam3+y] = tmp_sum + C[x*tam3+y];
    }
}

float codeGPU (float *A, float *B, float *C, float *D, int tam1,  int tam2, int tam3){
    
    // GPU-an non gordeko diren matrizeen espazioak reserbatu eta beharrezko datuak bidali
    float *d_A, *d_B, *d_C, *d_D;

    hipMalloc (&d_A, tam1 * tam2 * sizeof(float));
    hipMalloc (&d_B, tam2 * tam3 * sizeof(float));
    hipMalloc (&d_C, tam1 * tam3 * sizeof(float));
    hipMalloc (&d_D, tam1 * tam3 * sizeof(float));

    hipMemcpy (d_A, A, tam1 * tam2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy (d_B, B, tam2 * tam3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy (d_C, C, tam1 * tam3 * sizeof(float), hipMemcpyHostToDevice);

    //Denborari dagokion aldagaiak sortu eta hasieratu
    float Tex;
    hipEvent_t t0, t1;

    hipEventCreate(&t0);
    hipEventCreate(&t1);
    
    hipEventRecord(t0);

    //Funtzioari deitzeko datu egiturak sortu
    /*
    //2D
    dim3 threadsPerBlock (N, N); //dim3 threadsPerBlock (BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid (ceil((tam1*tam3)/N), ceil((tam1*tam3)/N));//dim3 blocksPerGrid (ceil(N/BLOCK_SIZE), ceil(N/BLOCK_SIZE));
  */  

    // 1D
    int threadsPerBlock = N; // Hari kopurua bloke bakoitzean, biren berretura eta 16 gutxienez
    int blocksPerGrid = (tam1 * tam3 + N - 1) / N; // Bloke kopurua, ondorioz, elementu kopurua zati hari kopurua goruntz borobilduta

    //Gure kernelaren exekuzioa 
    GPU_eragiketak<<<blocksPerGrid,  threadsPerBlock>>> (d_A, d_B, d_C, d_D, tam1, tam2, tam3);

    // Emaitza soilik itzuli GPU-tik
    hipMemcpy (D, d_D, tam1 * tam3 * sizeof(float), hipMemcpyDeviceToHost);

    //Bukaera den borak lortu eta hauen konparazioa Tex aldagaian gorde
    hipEventRecord(t1);
    hipEventSynchronize(t1);
    hipEventElapsedTime (&Tex, t0, t1);

    //Denbora aldagaiak liberatu
    hipEventDestroy(t0);
    hipEventDestroy(t1);

    //Matrizeak GPU-tik askatu
    hipFree (d_A);
    hipFree (d_B);
    hipFree (d_C);
    hipFree (d_D);

    return Tex;
}
